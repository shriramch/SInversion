#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

// Error checking for CUDA
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
template <typename T>
void check_cuda(T err, const char *const func, const char *const file,
                int const line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        exit(1);
    }
}

// Error checking for cuBLAS
#define checkCublasErrors(val) check_cublas((val), #val, __FILE__, __LINE__)
template <typename T>
void check_cublas(T err, const char *const func, const char *const file,
                  int const line) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS error at: " << file << ":" << line << std::endl;
        std::cerr << "Error code: " << err << " " << func << std::endl;
        exit(1);
    }
}

// Print the matrix
void printMatrix(const float *matrix, int width, int height) {
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            std::cout << matrix[i * width + j] << " ";
        }
        std::cout << std::endl;
    }
}

// Main function
int main() {
    const int N = 3; // Matrix size

    hipblasHandle_t handle;
    checkCublasErrors(hipblasCreate(&handle));

    // Allocate and initialize host matrices
    float A[N * N] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    float B[N * N] = {9, 8, 7, 6, 5, 4, 3, 2, 1};
    float C[N * N] = {0};

    float *d_A, *d_B, *d_C;
    checkCudaErrors(hipMalloc((void **)&d_A, N * N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_B, N * N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_C, N * N * sizeof(float)));

    // Copy matrices from the host to  device
    checkCudaErrors(
        hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(
        hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice));

    // Perform matrix multiplication: C = A * B
    const float alpha = 1.0f;
    const float beta = 0.0f;
    checkCublasErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                                  &alpha, d_A, N, d_B, N, &beta, d_C, N));

    // Copy  result back to  host
    checkCudaErrors(
        hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost));

    // Print  result
    std::cout << "Result Matrix C:" << std::endl;
    printMatrix(C, N, N);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    return 0;
}