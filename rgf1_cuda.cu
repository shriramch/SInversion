#include "hip/hip_runtime.h"
#include "argparse.h"
#include "rgf1.hpp"
#include "rgf1_cuda.hpp"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

float *identity_matrix;
int *d_info = nullptr;
float *d_A, *d_identity, *d_work;
int *ipiv;
float *d_result;

// void printFloatArray(const float arr[], int size) {
//     for (int i = 0; i < size; ++i) {
//         std::cout << arr[i] << " ";
//     }
//     std::cout << std::endl;
// }

// void printFloatArrayFromCuda(const float arr[], int size) {
//     float tempResult[size];
//     hipMemcpy(tempResult, arr, sizeof(float) * size,
//     hipMemcpyDeviceToHost); for (int i = 0; i < size; ++i) {
//         std::cout << tempResult[i] << " ";
//     }
//     std::cout << std::endl;
// }

void matrixMultiplyKernel(float *A, float *B, float *result, int n,
                          hipblasHandle_t cublasHandle) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, B, n,
                A, n, &beta, result, n);
}

__global__ void mulmul(float *A, float *B, float *result, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        result[row * n + col] = sum;
    }
}

__global__ void matrixSubtractKernel(float *A, float *B, float *result, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n * n) {
        result[index] = A[index] - B[index];
    }
}

__global__ void matrixAddKernel(float *A, float *B, float *result, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n * n) {
        result[index] = A[index] + B[index];
    }
}

__global__ void matrixScaleKernel(float *A, float k, float *result, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n * n) {
        result[index] = A[index] * k;
    }
}

// Function to create an identity matrix of size n x n
float *createIdentityMatrix(int n) {
    float *identityMatrix = (float *)malloc(n * n * sizeof(float));
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            int index = i * n + j;
            identityMatrix[index] = (i == j) ? 1 : 0;
        }
    }
    return identityMatrix;
}

void matrixInversionKernel(float *A, float *result, int n,
                           hipsolverHandle_t cusolverHandle) {

    // Copy the input matrix A to the device
    hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(d_identity, identity_matrix, n * n * sizeof(float),
               hipMemcpyHostToDevice);
    // hipMemcpy(result, identity_matrix, n * n * sizeof(float),
    //            hipMemcpyHostToDevice);

    // Perform LU decomposition on the device
    hipsolverDnSgetrf(cusolverHandle, n, n, d_A, n, d_work, NULL,
                     d_info); // Not using PIVOT for now

    // Solving AX = I  , where X is the result_matrix, and I is the
    // identity_matrix. Since AA^(-1) = I It saves on the result_matrix
    // (identity) the answer
    hipsolverDnSgetrs(cusolverHandle, HIPBLAS_OP_N, n, n, d_A, n, NULL,
                     d_identity, n, d_info); // Not using PIVOT for now

    // hipsolverDnSgetrs(cusolverHandle, HIPBLAS_OP_N, n, n, A, n, NULL,
    //                  result, n, d_info); // Not using PIVOT for now

    // std::cout << "printing d_identity from CUDA after hipsolverDnSgetrs: \n";
    // printFloatArrayFromCuda(d_identity, n * n);
    hipMemcpy(result, d_identity, n * n * sizeof(float),
               hipMemcpyDeviceToDevice);
}

void matrixTransposeKernel(const float *A, float *result, int n,
                           hipblasHandle_t cublasHandle) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Perform the transposition
    hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, A, n,
                &beta, NULL, n, result, n);
}

// CUDA-accelerated rgf1sided function
void rgf1sided_cuda(Matrix &input_A, Matrix &input_G, bool sym_mat,
                    bool save_off_diag) {
    int blockSize, matrixSize;
    input_A.getBlockSizeAndMatrixSize(blockSize, matrixSize);
    int nblocks = matrixSize / blockSize;

    int kernels_num_blocks = nblocks;
    int kernels_num_threads = nblocks;
    size_t blockSizeBytes = blockSize * blockSize * sizeof(float);

    // Initialize the handle used for cuBLAS
    hipblasHandle_t cublasHandle;
    hipsolverHandle_t cusolverHandle;

    hipblasCreate(&cublasHandle);
    hipsolverDnCreate(&cusolverHandle);

    // Allocate memory for Matrix specifics on the GPU
    size_t size_mdiag = nblocks * blockSize * blockSize * sizeof(float);
    size_t size_updiag = (nblocks - 1) * blockSize * blockSize * sizeof(float);
    float *A_mat, *A_mdiag, *G_mat, *G_mdiag;
    float *A_updiag, *G_updiag;
    float *A_lodiag, *G_lodiag;
    hipMalloc(&A_mat, size_mdiag + 2 * size_updiag + 6 * blockSizeBytes);
    hipMalloc(&G_mat, size_mdiag + 2 * size_updiag);
    A_mdiag = A_mat;
    A_updiag = A_mat + size_mdiag / sizeof(float);
    A_lodiag = A_updiag + size_updiag / sizeof(float);
    G_mdiag = G_mat;
    G_updiag = G_mat + size_mdiag / sizeof(float);
    G_lodiag = G_updiag + size_updiag / sizeof(float);

    // Copy matrices from host to device
    hipMemcpy(A_mdiag, input_A.mdiag, size_mdiag, hipMemcpyHostToDevice);
    hipMemcpy(A_updiag, input_A.updiag, size_updiag, hipMemcpyHostToDevice);
    hipMemcpy(A_lodiag, input_A.lodiag, size_updiag, hipMemcpyHostToDevice);

    // Utility matrices
    float *AAi, *AGi;
    AAi = A_lodiag + size_updiag / sizeof(float);
    AGi = AAi + blockSizeBytes / sizeof(float);
    float *Glf, *Glf1;
    Glf = AGi + blockSizeBytes / sizeof(float);
    Glf1 = Glf + blockSizeBytes / sizeof(float);
    float *Guf, *Guf1;
    Guf = Glf1 + blockSizeBytes / sizeof(float);
    Guf1 = Guf + blockSizeBytes / sizeof(float);

    // Inverse and transpose kernel variables
    hipMalloc(&d_info, sizeof(int));
    hipMalloc(&d_A, blockSize * blockSize * sizeof(float));
    hipMalloc(&d_identity, blockSize * blockSize * sizeof(float));
    hipMalloc(&d_work, blockSize * blockSize * sizeof(float));
    hipMalloc(&ipiv, blockSize * sizeof(int));
    hipMalloc((void **)&d_result, blockSize * blockSize * sizeof(float));

    identity_matrix = createIdentityMatrix(blockSize);

    // Launch CUDA kernels for matrix operations

    // 0. Inverse of the first block
    matrixInversionKernel(A_mdiag, G_mdiag, blockSize, cusolverHandle);
    // 1. Forward substitution (performed left to right)

    for (int i = 1; i < nblocks; ++i) {
        matrixMultiplyKernel(&(A_lodiag[(i - 1) * blockSize * blockSize]),
                             &(G_mdiag[(i - 1) * blockSize * blockSize]), AGi,
                             blockSize, cublasHandle);

        matrixMultiplyKernel(AGi, &(A_updiag[(i - 1) * blockSize * blockSize]),
                             AAi, blockSize, cublasHandle);

        matrixSubtractKernel<<<kernels_num_blocks, kernels_num_threads>>>(
            &(A_mdiag[i * blockSize * blockSize]), AAi, AGi, blockSize);

        matrixInversionKernel(AGi, &(G_mdiag[i * blockSize * blockSize]),
                              blockSize, cusolverHandle);
    }

    for (int i = nblocks - 2; i >= 0; --i) {
        matrixMultiplyKernel(&(G_mdiag[(i + 1) * blockSize * blockSize]),
                             &(A_lodiag[i * blockSize * blockSize]), Glf1,
                             blockSize, cublasHandle);
        matrixMultiplyKernel(Glf1, &(G_mdiag[i * blockSize * blockSize]), Glf,
                             blockSize, cublasHandle);

        if (save_off_diag) {
            matrixScaleKernel<<<kernels_num_blocks, kernels_num_threads>>>(
                Glf, -1, &(G_lodiag[i * blockSize * blockSize]), blockSize);

            if (sym_mat) {
                matrixTransposeKernel(&(G_lodiag[i * blockSize * blockSize]),
                                      &(G_updiag[i * blockSize * blockSize]),
                                      blockSize, cublasHandle);
            } else {

                matrixMultiplyKernel(
                    &(A_updiag[i * blockSize * blockSize]),
                    &(G_mdiag[(i + 1) * blockSize * blockSize]), Guf1,
                    blockSize, cublasHandle);
                matrixMultiplyKernel(&(G_mdiag[i * blockSize * blockSize]),
                                     Guf1, Guf, blockSize, cublasHandle);
                matrixScaleKernel<<<kernels_num_blocks, kernels_num_threads>>>(
                    Guf, -1, &(G_updiag[i * blockSize * blockSize]), blockSize);
            }
        }

        matrixMultiplyKernel(&(A_updiag[i * blockSize * blockSize]), Glf, Glf1,
                             blockSize, cublasHandle);
        matrixMultiplyKernel(&(G_mdiag[i * blockSize * blockSize]), Glf1, Glf,
                             blockSize, cublasHandle);
        matrixAddKernel<<<kernels_num_blocks, kernels_num_threads>>>(
            &(G_mdiag[i * blockSize * blockSize]), Glf,
            &(G_mdiag[i * blockSize * blockSize]), blockSize);
    }

    // Copy results back to host
    hipMemcpy(input_G.mdiag, G_mdiag, size_mdiag, hipMemcpyDeviceToHost);
    hipMemcpy(input_G.updiag, G_updiag, size_updiag, hipMemcpyDeviceToHost);
    hipMemcpy(input_G.lodiag, G_lodiag, size_updiag, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(A_mat);
    hipFree(G_mat);

    // Clean up of inverse kernel
    free(identity_matrix);
    hipFree(d_A);
    hipFree(d_work);
    hipFree(ipiv);
    hipFree(d_identity);
    hipFree(d_info);
    hipFree(d_result);

    // Destroy cuBLAS handle
    hipblasDestroy(cublasHandle);
    hipsolverDnDestroy(cusolverHandle);
}

// typedef struct {
//     int matrixSize;
//     int blockSize;
//     int numRuns;
//     bool isSymmetric;
//     bool saveOffDiag;
//     char *inputPath;
// } Config;

// void InitOptions(Config *config) {
//     config->blockSize = 2;
//     config->matrixSize = 0;
//     config->numRuns = 10;
//     config->isSymmetric = false;
//     config->saveOffDiag = true;
//     config->inputPath = NULL;
// }

// int parse(Config *config, int argc, const char **argv) {
//     static const char *const usages[] = {
//         NULL,
//     };
//     struct argparse_option options[] = {
//         OPT_HELP(),
//         OPT_INTEGER('m', "matrixSize", &config->matrixSize, "matrix size",
//         NULL,
//                     0, 0),
//         OPT_INTEGER('b', "blockSize", &config->blockSize, "block size", NULL,
//         0,
//                     0),
//         OPT_INTEGER('n', "numRuns", &config->numRuns, "number of runs", NULL,
//         0,
//                     0),
//         OPT_INTEGER('s', "isSymmetric", &config->isSymmetric, "is symmetric",
//                     NULL, 0, 0),
//         OPT_INTEGER('o', "saveOffDiag", &config->saveOffDiag, "save off diag", NULL, 0, 0), OPT_STRING('f', "inputPath", &config->inputPath, "input path", NULL,0,0), OPT_END(),
//     };

//     struct argparse argparse;
//     argparse_init(&argparse, options, usages, 0);
//     argparse_describe(&argparse, "DPHPC TEAM", NULL);
//     argc = argparse_parse(&argparse, argc, argv);

//     return 0;
// }

// int main(int argc, const char *argv[]) {
//     // const char *bin_name = argv[0];
//     Config config;
//     InitOptions(&config);
//     parse(&config, argc, argv);
//     if (config.inputPath != NULL) {
//         // read matrix from file
//     } else if (config.matrixSize != 0) {
//         // generate matrix
//         int MATRIX_SIZE = config.matrixSize;
//         int BLOCK_SIZE = config.blockSize;
//         assert(MATRIX_SIZE % BLOCK_SIZE == 0);
//         // int NUM_RUNS = config.numRuns;
//         bool IS_SYMMETRIC = config.isSymmetric;
//         bool SAVE_OFF_DIAG = config.saveOffDiag;

//         Matrix inputMatrix =
//             generateBandedDiagonalMatrix(MATRIX_SIZE, 2, true, 0);

//         // Matrix inputMatrix = generateFixedMatrixOfSize4();
//         // inputMatrix.convertDenseToBlkTridiag(BLOCK_SIZE);

//         // inputMatrix.printB();
//         Matrix tempResult(
//             MATRIX_SIZE); // zero initialization, same shape as inputMatrix
//         tempResult.convertDenseToBlkTridiag(
//             BLOCK_SIZE); // G has same blockSize as inputMatrix
//         rgf1sided_cuda(inputMatrix, tempResult, IS_SYMMETRIC, SAVE_OFF_DIAG);

//         tempResult.printB();
//         std::cout << "\n########################################## \n";

//         // inputMatrix.printB();
//         // Check against the already implemented RGF1 on C++
//         Matrix tempResult_cpp(
//             MATRIX_SIZE); // zero initialization, same shape as inputMatrix
//         tempResult_cpp.convertDenseToBlkTridiag(
//             BLOCK_SIZE); // G has same blockSize as inputMatrix

//         rgf1sided(inputMatrix, tempResult_cpp, IS_SYMMETRIC, SAVE_OFF_DIAG);

//         tempResult_cpp.printB();

//     }
// }