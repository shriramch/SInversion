#include "hip/hip_runtime.h"
#include "argparse.h"
#include "rgf1.hpp"
#include "rgf1_cuda.hpp"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

void printFloatArray(const float arr[], int size) {
    // std::cout << "Array of floats: \n";
    for (int i = 0; i < size; ++i) {
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;
}

void printFloatArrayFromCuda(const float arr[], int size) {
    float tempResult[size];
    hipMemcpy(tempResult, arr, sizeof(float) * size, hipMemcpyDeviceToHost);
    // std::cout << "Array of floats from GPU: \n";
    for (int i = 0; i < size; ++i) {
        std::cout << tempResult[i] << " ";
    }
    std::cout << std::endl;
}

void matrixMultiplyKernel(float *A, float *B, float *result, int n,
                          hipblasHandle_t cublasHandle) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, B, n,
                A, n, &beta, result, n);
}

__global__ void mulmul(float *A, float *B, float *result, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        result[row * n + col] = sum;
    }
}

__global__ void matrixSubtractKernel(float *A, float *B, float *result, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n * n) {
        result[index] = A[index] - B[index];
    }
}

__global__ void matrixAddKernel(float *A, float *B, float *result, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n * n) {
        result[index] = A[index] + B[index];
    }
}

__global__ void matrixScaleKernel(float *A, float k, float *result, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n * n) {
        result[index] = A[index] * k;
    }
}

// Function to create an identity matrix of size n x n
float *createIdentityMatrix(int n) {
    float *identityMatrix = (float *)malloc(n * n * sizeof(float));
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            int index = i * n + j;
            identityMatrix[index] = (i == j) ? 1 : 0;
        }
    }
    return identityMatrix;
}

void matrixInversionKernel(float *A, float *result, int n,
                           hipsolverHandle_t cusolverHandle) {
    float *identity_matrix = createIdentityMatrix(n);
    int *d_info = nullptr; /* error info */
    hipMalloc(&d_info, sizeof(int));

    // Create a temporary matrix on the device
    float *d_A, *d_identity, *d_work;
    hipMalloc(&d_A, n * n * sizeof(float));
    hipMalloc(&d_identity, n * n * sizeof(float));
    hipMalloc(&d_work, n * n * sizeof(float));
    int *ipiv;
    hipMalloc(&ipiv, n * sizeof(int));

    // Copy the input matrix A to the device
    hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_identity, identity_matrix, n * n * sizeof(float),
               hipMemcpyHostToDevice);

    // Perform LU decomposition on the device
    hipsolverDnSgetrf(cusolverHandle, n, n, d_A, n, d_work, NULL,
                     d_info); // Not using PIVOT for now

    // Solving AX = I  , where X is the result_matrix, and I is the
    // identity_matrix. Since AA^(-1) = I It saves on the result_matrix
    // (identity) the answer
    hipsolverDnSgetrs(cusolverHandle, HIPBLAS_OP_N, n, n, d_A, n, NULL,
                     d_identity, n, d_info); // Not using PIVOT for now

    // std::cout << "printing d_identity from CUDA after hipsolverDnSgetrs: \n";
    // printFloatArrayFromCuda(d_identity, n * n);
    hipMemcpy(result, d_identity, n * n * sizeof(float),
               hipMemcpyDeviceToHost);

    // Clean up
    free(identity_matrix);
    hipFree(d_A);
    hipFree(d_work);
    hipFree(ipiv);
    hipFree(d_identity);
    hipFree(d_info);
}

void matrixTransposeKernel(const float *A, float *result, int n,
                           hipblasHandle_t cublasHandle) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Allocate device memory for the input and output matrices
    float *d_A, *d_result;
    hipMalloc((void **)&d_A, n * n * sizeof(float));
    hipMalloc((void **)&d_result, n * n * sizeof(float));

    // Copy the input matrix A to device memory
    hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);

    // Perform the transposition
    hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, d_A, n,
                &beta, NULL, n, d_result, n);

    // Copy the transposed matrix back to the host memory
    hipMemcpy(result, d_result, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_result);
}

// CUDA-accelerated rgf1sided function
void rgf1sided_cuda(Matrix &input_A, Matrix &input_G, bool sym_mat,
                    bool save_off_diag) {
    int blockSize, matrixSize;
    input_A.getBlockSizeAndMatrixSize(blockSize, matrixSize);
    int nblocks = matrixSize / blockSize;

    // Initialize the handle used for cuBLAS
    hipblasHandle_t cublasHandle;
    hipsolverHandle_t cusolverHandle;

    hipblasCreate(&cublasHandle);
    hipsolverDnCreate(&cusolverHandle);

    // Allocate memory for matrices on the GPU
    float *A, *G;
    size_t size = matrixSize * matrixSize * sizeof(float);
    int matrix_array_size = matrixSize * matrixSize;
    hipMalloc(&A, size);
    hipMalloc(&G, size);

    // Copy matrices from host to device
    hipMemcpy(A, input_A.getMat(), size, hipMemcpyHostToDevice);
    hipMemcpy(G, input_G.getMat(), size, hipMemcpyHostToDevice);

    // Allocate memory for Matrix specifics on the GPU
    float *A_mdiag, *G_mdiag;
    size_t size_mdiag = nblocks * blockSize * blockSize * sizeof(float);
    hipMalloc(&A_mdiag, size_mdiag);
    hipMalloc(&G_mdiag, size_mdiag);

    // Copy matrices from host to device
    hipMemcpy(A_mdiag, input_A.mdiag, size_mdiag, hipMemcpyHostToDevice);
    hipMemcpy(G_mdiag, input_G.mdiag, size_mdiag, hipMemcpyHostToDevice);

    float *A_updiag, *G_updiag;
    size_t size_updiag = (nblocks - 1) * blockSize * blockSize * sizeof(float);
    hipMalloc(&A_updiag, size_updiag);
    hipMalloc(&G_updiag, size_updiag);

    // Copy matrices from host to device
    hipMemcpy(A_updiag, input_A.updiag, size_updiag, hipMemcpyHostToDevice);
    hipMemcpy(G_updiag, input_G.updiag, size_updiag, hipMemcpyHostToDevice);

    float *A_lodiag, *G_lodiag;
    hipMalloc(&A_lodiag, size_updiag);
    hipMalloc(&G_lodiag, size_updiag);

    // Copy matrices from host to device
    hipMemcpy(A_lodiag, input_A.lodiag, size_updiag, hipMemcpyHostToDevice);
    hipMemcpy(G_lodiag, input_G.lodiag, size_updiag, hipMemcpyHostToDevice);

    // Launch CUDA kernels for matrix operations

    // 0. Inverse of the first block
    matrixInversionKernel(A_mdiag, G_mdiag, blockSize, cusolverHandle);

    int kernels_num_blocks = nblocks;
    int kernels_num_threads = nblocks;
    size_t blockSizeBytes = blockSize * blockSize * sizeof(float);
    // 1. Forward substitution (performed left to right)

    for (int i = 1; i < nblocks; ++i) {
        float *AAi, *AGi;

        hipMalloc(&AAi, blockSizeBytes);
        hipMalloc(&AGi, blockSizeBytes);

        hipMemset(AAi, 0, blockSizeBytes);
        hipMemset(AGi, 0, blockSizeBytes);

        // TODO, check how to parallelize, since u need the previous G
        matrixMultiplyKernel(&(A_lodiag[(i - 1) * blockSize * blockSize]),
                             &(G_mdiag[(i - 1) * blockSize * blockSize]), AGi,
                             blockSize, cublasHandle);
        matrixMultiplyKernel(AGi, &(A_updiag[(i - 1) * blockSize * blockSize]),
                             AAi, blockSize, cublasHandle);
        matrixSubtractKernel<<<kernels_num_blocks, kernels_num_threads>>>(
            &(A_mdiag[i * blockSize * blockSize]), AAi, AGi, blockSize);
        matrixInversionKernel(AGi, &(G_mdiag[i * blockSize * blockSize]),
                              blockSize, cusolverHandle);

        // Free temporary GPU memory
        hipFree(AAi);
        hipFree(AGi);
    }

    // 2. Backward substitution
    float *Glf, *Glf1;
    hipMalloc(&Glf, blockSizeBytes);
    hipMalloc(&Glf1, blockSizeBytes);

    for (int i = nblocks - 2; i >= 0; --i) {
        matrixMultiplyKernel(&(G_mdiag[(i + 1) * blockSize * blockSize]),
                             &(A_lodiag[i * blockSize * blockSize]), Glf1,
                             blockSize, cublasHandle);
        matrixMultiplyKernel(Glf1, &(G_mdiag[i * blockSize * blockSize]), Glf,
                             blockSize, cublasHandle);

        if (save_off_diag) {
            matrixScaleKernel<<<kernels_num_blocks, kernels_num_threads>>>(
                Glf, -1, &(G_lodiag[i * blockSize * blockSize]), blockSize);

            if (sym_mat) {
                matrixTransposeKernel(&(G_lodiag[i * blockSize * blockSize]),
                                      &(G_updiag[i * blockSize * blockSize]),
                                      blockSize, cublasHandle);
            } else {
                float *Guf, *Guf1;
                hipMalloc(&Guf, blockSizeBytes);
                hipMalloc(&Guf1, blockSizeBytes);

                matrixMultiplyKernel(
                    &(A_updiag[i * blockSize * blockSize]),
                    &(G_mdiag[(i + 1) * blockSize * blockSize]), Guf1,
                    blockSize, cublasHandle);
                matrixMultiplyKernel(&(G_mdiag[i * blockSize * blockSize]),
                                     Guf1, Guf, blockSize, cublasHandle);
                matrixScaleKernel<<<kernels_num_blocks, kernels_num_threads>>>(
                    Guf, -1, &(G_updiag[i * blockSize * blockSize]), blockSize);

                // Free temporary GPU memory
                hipFree(Guf);
                hipFree(Guf1);
            }
        }

        matrixMultiplyKernel(&(A_updiag[i * blockSize * blockSize]), Glf, Glf1,
                             blockSize, cublasHandle);
        matrixMultiplyKernel(&(G_mdiag[i * blockSize * blockSize]), Glf1, Glf,
                             blockSize, cublasHandle);
        matrixAddKernel<<<kernels_num_blocks, kernels_num_threads>>>(
            &(G_mdiag[i * blockSize * blockSize]), Glf,
            &(G_mdiag[i * blockSize * blockSize]), blockSize);
    }

    // Free temporary GPU memory
    hipFree(Glf);
    hipFree(Glf1);

    // printFloatArrayFromCuda(G, matrix_array_size);

    // Copy results back to host
    hipMemcpy(input_A.getMat(), A, size, hipMemcpyDeviceToHost);
    hipMemcpy(input_G.getMat(), G, size, hipMemcpyDeviceToHost);
    hipMemcpy(input_A.mdiag, A_mdiag, size_mdiag, hipMemcpyDeviceToHost);
    hipMemcpy(input_G.mdiag, G_mdiag, size_mdiag, hipMemcpyDeviceToHost);
    hipMemcpy(input_A.updiag, A_updiag, size_updiag, hipMemcpyDeviceToHost);
    hipMemcpy(input_G.updiag, G_updiag, size_updiag, hipMemcpyDeviceToHost);
    hipMemcpy(input_A.lodiag, A_lodiag, size_updiag, hipMemcpyDeviceToHost);
    hipMemcpy(input_G.lodiag, G_lodiag, size_updiag, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(A);
    hipFree(G);
    hipFree(A_mdiag);
    hipFree(G_mdiag);
    hipFree(A_updiag);
    hipFree(G_updiag);
    hipFree(A_lodiag);
    hipFree(G_lodiag);

    // Destroy cuBLAS handle
    hipblasDestroy(cublasHandle);
    hipsolverDnDestroy(cusolverHandle);
}

// int main(int argc, const char *argv[]) {
//     const char *bin_name = argv[0];
//     Config config;
//     InitOptions(&config);
//     parse(&config, argc, argv);
//     if (config.inputPath != NULL) {
//         // read matrix from file
//     } else if (config.matrixSize != 0) {
//         // generate matrix
//         int MATRIX_SIZE = config.matrixSize;
//         int BLOCK_SIZE = config.blockSize;
//         assert(MATRIX_SIZE % BLOCK_SIZE == 0);
//         int NUM_RUNS = config.numRuns;
//         bool IS_SYMMETRIC = config.isSymmetric;
//         bool SAVE_OFF_DIAG = config.saveOffDiag;

//         Matrix inputMatrix =
//             generateBandedDiagonalMatrix(MATRIX_SIZE, 2, true, 0);

//         // Matrix inputMatrix = generateFixedMatrixOfSize4();
//         inputMatrix.convertDenseToBlkTridiag(BLOCK_SIZE);

//         // inputMatrix.printB();
//         Matrix tempResult(
//             MATRIX_SIZE); // zero initialization, same shape as inputMatrix
//         tempResult.convertDenseToBlkTridiag(
//             BLOCK_SIZE); // G has same blockSize as inputMatrix
//         rgf1sided_cuda(inputMatrix, tempResult, IS_SYMMETRIC, SAVE_OFF_DIAG);

//         tempResult.printB();
//         std::cout << "\n########################################## \n";

//         // inputMatrix.printB();
//         // Check against the already implemented RGF1 on C++
//         Matrix tempResult_cpp(
//             MATRIX_SIZE); // zero initialization, same shape as inputMatrix
//         tempResult_cpp.convertDenseToBlkTridiag(
//             BLOCK_SIZE); // G has same blockSize as inputMatrix

//         rgf1sided(inputMatrix, tempResult_cpp, IS_SYMMETRIC, SAVE_OFF_DIAG);

//         tempResult_cpp.printB();
//     }
// }