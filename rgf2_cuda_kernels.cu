#include "hip/hip_runtime.h"
#include "rgf2_cuda.hpp"
// #include <hipblas.h>
#include <hip/hip_runtime.h>
// #include <hipsolver.h>
#include <hip/hip_runtime.h>

int kernels_num_blocks, kernels_num_threads;

void kernel_init(int n) {
  kernels_num_blocks = kernels_num_threads = n;
}

__global__ void matrixSubtractKernel(float *A, float *B, float *result, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n * n) {
        result[index] = A[index] - B[index];
    }
}

void matrixSubtracter(float *A, float *B, float *result, int n) {
  matrixSubtractKernel<<<kernels_num_blocks, kernels_num_threads>>>(A, B, result, n);
}

__global__ void matrixAddKernel(float *A, float *B, float *result, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n * n) {
        result[index] = A[index] + B[index];
    }
}

void matrixAdder(float *A, float *B, float *result, int n) {
  matrixAddKernel<<<kernels_num_blocks, kernels_num_threads>>>(A, B, result, n);
}

__global__ void matrixScaleKernel(float *A, float k, float *result, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n * n) {
        result[index] = A[index] * k;
    }
}

void matrixScaler(float *A, float k, float *result, int n) {
  matrixScaleKernel<<<kernels_num_blocks, kernels_num_threads>>>(A, k, result, n);
}

// void matrixMultiplyKernel(float *A, float *B, float *result, int n,
//                           hipblasHandle_t cublasHandle) {
//     const float alpha = 1.0f;
//     const float beta = 0.0f;
//     hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n,
//                 B, n, &beta, result, n);
// }

// void matrixInversionKernel(float *A, float *result, int n,
//                            hipsolverHandle_t cusolverHandle) {
//     float *identity_matrix = createIdentityMatrix(n);
//     int *d_info = nullptr; /* error info */
//     hipMalloc(&d_info, sizeof(int));

//     // Create a temporary matrix on the device
//     float *d_A, *d_identity, *d_work;
//     hipMalloc(&d_A, n * n * sizeof(float));
//     hipMalloc(&d_identity, n * n * sizeof(float));
//     hipMalloc(&d_work, n * n * sizeof(float));
//     int *ipiv;
//     hipMalloc(&ipiv, n * sizeof(int));

//     // Copy the input matrix A to the device
//     hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);
//     hipMemcpy(d_identity, identity_matrix, n * n * sizeof(float),
//                hipMemcpyHostToDevice);

//     // Perform LU decomposition on the device
//     hipsolverDnSgetrf(cusolverHandle, n, n, d_A, n, d_work, NULL,
//                      d_info); // Not using PIVOT for now

//     // Solving AX = I  , where X is the result_matrix, and I is the
//     // identity_matrix. Since AA^(-1) = I It saves on the result_matrix
//     // (identity) the answer
//     hipsolverDnSgetrs(cusolverHandle, HIPBLAS_OP_N, n, n, d_A, n, NULL,
//                      d_identity, n, d_info); // Not using PIVOT for now

//     // std::cout << "printing d_identity from CUDA after hipsolverDnSgetrs: \n";
//     // printFloatArrayFromCuda(d_identity, n * n);
//     hipMemcpy(result, d_identity, n * n * sizeof(float),
//                hipMemcpyDeviceToHost);

//     // Clean up
//     free(identity_matrix);
//     hipFree(d_A);
//     hipFree(d_work);
//     hipFree(ipiv);
//     hipFree(d_identity);
//     hipFree(d_info);
// }

// void matrixTransposeKernel(const float *A, float *result, int n,
//                            hipblasHandle_t cublasHandle) {
//     const float alpha = 1.0f;
//     const float beta = 0.0f;

//     // Allocate device memory for the input and output matrices
//     float *d_A, *d_result;
//     hipMalloc((void **)&d_A, n * n * sizeof(float));
//     hipMalloc((void **)&d_result, n * n * sizeof(float));

//     // Copy the input matrix A to device memory
//     hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);

//     // Perform the transposition
//     hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, d_A, n,
//                 &beta, NULL, n, d_result, n);

//     // Copy the transposed matrix back to the host memory
//     hipMemcpy(result, d_result, n * n * sizeof(float), hipMemcpyDeviceToHost);

//     // Free device memory
//     hipFree(d_A);
//     hipFree(d_result);
// }

// void rgf2sided_upperprocess_cuda_prempi(Matrix &input_A, Matrix &input_G, int nblocks_2,
//                                  bool sym_mat,
//                                  bool save_off_diag, float *send);

// void rgf2sided_upperprocess_cuda_postmpi(Matrix &input_A, Matrix &input_G, int nblocks_2,
//                                  bool sym_mat,
//                                  bool save_off_diag, float *recv);

// void rgf2sided_lowerprocess_cuda_prempi(Matrix &input_A, Matrix &input_G, int nblocks_2,
//                                  bool sym_mat,
//                                  bool save_off_diag);

// void rgf2sided_lowerprocess_cuda_midmpi(Matrix &input_A, Matrix &input_G, int nblocks_2,
//                                  bool sym_mat,
//                                  bool save_off_diag);

// void rgf2sided_lowerprocess_cuda_postmpi(Matrix &input_A, Matrix &input_G, int nblocks_2,
//                                  bool sym_mat,
//                                  bool save_off_diag);