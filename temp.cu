
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <iostream>

class Matrix {
public:
    static void invCUDA(int n, const float *A, float *result);
};

int main() {
    const int n = 4;
    const float A[] = {1, 0, 0, 0, 0, 2, 0, 0, 0, 0, 3, 0, 0, 0, 0, 4};
    float result[n * n];

    Matrix::invCUDA(n, A, result);

    // Print the result
    std::cout << "Inverse Matrix:" << std::endl;
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << result[i * n + j] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}

void Matrix::invCUDA(int n, const float *A, float *result) {
    hipsolverHandle_t handle;
    hipsolverDnCreate(&handle);

    // Create a temporary matrix on the device
    float *d_A, *d_result;
    hipMalloc(&d_A, n * n * sizeof(float));
    hipMalloc(&d_result, n * n * sizeof(float));

    // Copy the input matrix A to the device
    hipMemcpy(d_A, A, n * n * sizeof(float), hipMemcpyHostToDevice);

    // Perform LU decomposition on the device
    int *ipiv;
    hipMalloc(&ipiv, n * sizeof(int));
    hipsolverDnSgetrf(handle, n, n, d_A, n, d_result, ipiv, nullptr);

    // Solve for each column of the identity matrix to obtain the inverse
    float *d_identity;
    hipMalloc(&d_identity, n * sizeof(float));

    for (int i = 0; i < n; ++i) {
        hipMemcpy(d_identity, &A[i], n * sizeof(float), hipMemcpyHostToDevice);
        hipsolverDnSgetrs(handle, HIPBLAS_OP_N, n, 1, d_result, n, ipiv, d_identity, n, nullptr);
        hipMemcpy(&result[i * n], d_identity, n * sizeof(float), hipMemcpyDeviceToHost);
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_result);
    hipFree(ipiv);
    hipFree(d_identity);

    hipsolverDnDestroy(handle);
}
